#include <stdio.h>
#include "problem.h"
#include "twoPhaseMethod.h"
#include "macro.h"
#include "time.h"

void setupDevice();

int main(int argc, const char *argv[])
{
    printf("Starting....\n");

    setupDevice();

    FILE *file = NULL;
    int vars;
    int constraints;
    problem_t *problem;

    if (argc == 2)
    {
        printf("Leggo problema da file\n");
        if (fopen_s(&file, argv[1], "r") != 0)
        {
            fprintf(stderr, "Errore nell'apertura del file");
            exit(-1);
        }

        problem = readProblemFromFile(file);
        fclose(file);
    }
    else if (argc == 3)
    {
        printf("Genero problema casuale\n");
        vars = atoi(argv[1]);
        constraints = atoi(argv[2]);

        problem = generateRandomProblem(vars, constraints, 0);
    }
    else
    {
        fprintf_s(stderr, "Argomenti mancanti!");
        exit(-1);
    }

#ifdef DEBUG
    printProblemToStream(stdout, problem);
#endif

    TYPE *solution = (TYPE *)(malloc(BYTE_SIZE(problem->vars)));
    TYPE optimalValue = 0;
    FILE *fileSolution = NULL;
    if (fopen_s(&file, "solution.txt", "w") != 0)
    {
        fprintf(stderr, "Errore nell'apertura del file");
        exit(-1);
    }

    printf("Resolving....\n");
    switch (twoPhaseMethod(problem, solution, &optimalValue))
    {
    case INFEASIBLE:
        printf("Problem INFEASIBLE!\n");
        break;

    case UNBOUNDED:
        printf("Problem UNBOUNDED!\n");
        break;

    case DEGENERATE:
        printf("Problem DEGENERATE!\n");
        break;

    default:
        printf("Problem solved!\n");

        for (size_t i = 0; i < problem->vars; i++)
        {
            fprintf_s(fileSolution, "%lf\n", solution[i]);
        }
        fprintf_s(fileSolution, "\nOptimal value: %lf\n", optimalValue);
        break;
    }
    fclose(fileSolution);
    
    free(solution);
    freeProblem(problem);
}

void setupDevice()
{
    if (TYPE_SIZE == 8)
    {
        hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
        printf("Bank size set to 8 byte\n");
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    if (!prop.canMapHostMemory)
    {
        fprintf_s(stderr, "Device cannot map memory!\n");
        exit(-1);
    }
    hipSetDeviceFlags(hipDeviceMapHost);
}
