#include "tabular.cuh"
#include "error.cuh"

tabular_t* newTabular(problem_t* problem)
{
    tabular_t* tabular = (tabular_t*)malloc(sizeof(tabular_t));
    
    tabular->problem = problem;
    //Registro i vettori del problema come memoria page-locked (per poter utilizzare i trasferimenti paralleli con gli stream)
    HANDLE_ERROR(hipHostRegister(problem->constraintsMatrix, BYTE_SIZE(problem->vars * problem->constraints), hipHostRegisterDefault));
    HANDLE_ERROR(hipHostRegister(problem->knownTermsVector, BYTE_SIZE(problem->constraints), hipHostRegisterDefault));
    HANDLE_ERROR(hipHostRegister(problem->objectiveFunction, BYTE_SIZE(problem->vars), hipHostRegisterDefault));

    tabular->rows = problem->constraints + 1;
    tabular->pitch = 0;
    tabular->cols = 0;
    tabular->table = NULL;
    tabular->indicatorCol = NULL;
    tabular->r0 = NULL;

    tabular->base = (int*)malloc(BYTE_SIZE(tabular->rows));
    memset(tabular->base, 0, BYTE_SIZE(tabular->rows));

    return tabular;
}

void print(FILE* Stream, tabular_t* tabular)
{
    TYPE* hTable = (TYPE*)malloc(BYTE_SIZE(tabular->rows * tabular->cols));
    TYPE* hIndicators = (TYPE*)malloc(BYTE_SIZE(tabular->cols));

    HANDLE_ERROR(hipMemcpy2D(
        hTable,
        BYTE_SIZE(tabular->cols),
        tabular->table,
        tabular->pitch,
        BYTE_SIZE(tabular->cols),
        tabular->rows,
        hipMemcpyDeviceToHost
    ));

    HANDLE_ERROR(hipMemcpy(
        hIndicators,
        tabular->indicatorCol,
        tabular->rows,
        hipMemcpyDeviceToHost
    ));

    fprintf(Stream, "\n--------------- Tabular --------------\n");
    for (size_t i = 0; i < tabular->rows; i++)
    {
        for (size_t j = 0; j < tabular->cols; j++)
        {
            fprintf(Stream, "%.2lf\t", hTable[i * tabular->cols + j]);
        }
        fprintf(Stream, "%.2lf\n", hIndicators[i]);
    }
    fprintf(Stream, "\n--------------------------------------\n");
    fprintf(Stream, "Base: ");
    for (size_t i = 1; i < tabular->rows; i++)
    {
        fprintf(Stream, "%.2lf\t", tabular->base[i]);
    }
    fprintf(Stream, "\n--------------------------------------\n");
}

void printTableauToStream(FILE* Stream, tabular_t* tabular)
{
    if (tabular->table != NULL)
    {
        print(Stream, tabular);
    }
}

void freeTabular(tabular_t* tabular)
{
    HANDLE_ERROR(hipHostUnregister(tabular->problem->constraintsMatrix));
    HANDLE_ERROR(hipHostUnregister(tabular->problem->knownTermsVector));
    HANDLE_ERROR(hipHostUnregister(tabular->problem->objectiveFunction));

    if (tabular->table != NULL)
    {
        HANDLE_ERROR(hipFree(tabular->table));
        HANDLE_ERROR(hipFree(tabular->indicatorCol));
        HANDLE_ERROR(hipFree(tabular->r0));
    }

    free(tabular->base);
    free(tabular);
}