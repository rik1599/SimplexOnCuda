#include "hip/hip_runtime.h"
#include "reduction.cuh"
#include "error.cuh"

#define THREADS 512
#define BL(N) min((N + THREADS - 1) / THREADS, 1024)

// ============ minElement ====================
__inline__ __device__ void warpReduceMin(volatile TYPE *pVal, volatile int *pIndex)
{
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1)
    {
        TYPE shflVal = __shfl_down_sync(warpSize - 1, *pVal, offset);
        int shfIndex = __shfl_down_sync(warpSize - 1, *pIndex, offset);
        if (shflVal < *pVal)
        {
            *pVal = shflVal;
            *pIndex = shfIndex;
        }
    }
}

__inline__ __device__ void blockReduceMin(volatile TYPE *pVal, volatile int *pIndex)
{
    static __shared__ TYPE sdata[32];
    static __shared__ int sindex[32];

    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;

    warpReduceMin(pVal, pIndex);

    if (lane == 0)
    {
        sdata[wid] = *pVal;
        sindex[wid] = *pIndex;
    }

    __syncthreads();

    *pVal = (threadIdx.x < blockDim.x / warpSize) ? sdata[lane] : INT_MAX;
    *pIndex = (threadIdx.x < blockDim.x / warpSize) ? sindex[lane] : -1;

    if (wid == 0)
    {
        warpReduceMin(pVal, pIndex);
    }
}

template <bool isFirstExecution>
__global__ void deviceReduceKernel(TYPE* g_values, unsigned int* g_index, int N)
{
    TYPE minVal = INT_MAX * 1.0;
    int minIndex = -1;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
        i < N; 
        i += blockDim.x * gridDim.x
    )
    {
        TYPE candidate = g_values[i];
        if (candidate < minVal)
        {
            minVal = candidate;

            if (isFirstExecution)
                minIndex = i;
            else
                minIndex = g_index[i];
        }
    }

    blockReduceMin(&minVal, &minIndex);

    if (threadIdx.x == 0)
    {
        g_values[blockIdx.x] = minVal;
        g_index[blockIdx.x] = minIndex;
    }
}

TYPE minElement(TYPE* g_vet, unsigned int size, unsigned int* outIndex)
{
    unsigned int* g_index;
    HANDLE_ERROR(hipMalloc((void**)&g_index, BL(size) * sizeof(unsigned int)));

    deviceReduceKernel<true><<<BL(size), THREADS>>>(g_vet, g_index, size);
    if (BL(size) > 1)
    {
        deviceReduceKernel<false><<<1, 1024>>>(g_vet, g_index, BL(size));
    }

    TYPE parallelMin;
    HANDLE_ERROR(hipMemcpy(&parallelMin, g_vet, sizeof(int), hipMemcpyDefault));
    HANDLE_ERROR(hipMemcpy(outIndex, g_index, sizeof(int), hipMemcpyDefault));

    hipFree(g_index);

    return parallelMin;
}

// ============ reduction with atomic ====================
template <bool minimum>
__inline__ __device__ void warpReduce(volatile TYPE *pVal)
{
    for (int offset = warpSize/2; offset > 0; offset /= 2)
    {   
        if (minimum)
            *pVal = min(*pVal, __shfl_down_sync(warpSize - 1, *pVal, offset));
        else
            *pVal = max(*pVal, __shfl_down_sync(warpSize - 1, *pVal, offset));
    }
}

template <bool minimum>
__inline__ __device__ void blockReduce(volatile TYPE *pVal)
{
    static __shared__ TYPE sdata[32];

    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;

    warpReduce<minimum>(pVal);

    if (lane == 0)
    {
        sdata[wid] = *pVal;
    }

    __syncthreads();

    if (minimum)
        *pVal = (threadIdx.x < blockDim.x / warpSize) ? sdata[lane] : INT_MAX;
    else
        *pVal = (threadIdx.x < blockDim.x / warpSize) ? sdata[lane] : INT_MIN;

    if (wid == 0)
    {
        warpReduce<minimum>(pVal);
    }
}

template <bool minimum>
__global__ void deviceReduceBlockAtomicKernel(TYPE* g_data, unsigned int N)
{
    TYPE partial = minimum ? INT_MAX : INT_MIN;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
        i < N; 
        i += blockDim.x * gridDim.x
    )
    {
        if (minimum)
            partial = min(partial, g_data[i]);
        else
            partial = max(partial, g_data[i]);
    }

    blockReduce<minimum>(&partial);

    if (threadIdx.x == 0)
    {
        if (minimum)
            atomicMin(g_data, partial);
        else
            atomicMax(g_data, partial);
    }
}

bool isGreaterThanZero(TYPE* g_vet, unsigned int size)
{
    return true;
}

bool isLessThanZero(TYPE* g_vet, unsigned int size)
{
    TYPE* g_vetCpy;
    HANDLE_ERROR(hipMalloc((void**)&g_vetCpy, BYTE_SIZE(size)));
    HANDLE_ERROR(hipMemcpy(g_vetCpy, g_vet, size, hipMemcpyDefault));

    deviceReduceBlockAtomicKernel<false><<<BL(size), THREADS>>>(g_vetCpy, size);
    TYPE maximum = *g_vetCpy;
    HANDLE_ERROR(hipFree(g_vetCpy));
    
    return (maximum <= 0);
}