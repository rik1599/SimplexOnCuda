#include "hip/hip_runtime.h"
#include "twoPhaseMethod.h"
#include "error.cuh"
#include "gaussian.cuh"

#define THREADS 512
#define BL(N) min((N + THREADS - 1) / THREADS, 1024)

/** Inserisce due matrici di indentità in coda a una matrice
 *  Si suppone sia linearizzata per colonne (non penso sia possibile generalizzare)
 *
 * @param mat - puntatore alla matrice
 * @param cols - colonne della matrice
 * @param pitch - il pitch della matrice
 */
__global__ void fillMatrix(TYPE *mat, int cols, size_t pitch)
{
    for (int idX = threadIdx.x + blockIdx.x * blockDim.x;
         idX < cols;
         idX += gridDim.x * blockDim.x)
    {
        *(INDEX(mat, idX, idX, pitch)) = 1;
        *(INDEX(mat, idX + cols, idX, pitch)) = 1;
    }
}

/**
 * Inizializza il vettore della base con numeri progressivi da start a start + size - 1
 */
__global__ void fillBaseVector(int *base, int size, int start)
{
    for (int idX = threadIdx.x + blockIdx.x * blockDim.x;
         idX < size;
         idX += gridDim.x * blockDim.x)
    {
        base[idX] = (start + idX);
    }
}

/**
 * Setta a 1 tutti gli elementi del vettore da start alla fine del vettore
 *
 * @param vector - puntatore al vettore da settare ad 1
 * @param size - la dimensione del vettore
 */
__global__ void setVectorToOne(TYPE *vector, int size)
{
    for (int idX = threadIdx.x + blockIdx.x * blockDim.x;
         idX < size;
         idX += gridDim.x * blockDim.x)
    {
        vector[idX] = 1;
    }
}

/**
 * Inverte i segni a tutti gli elementi del vettore da start alla fine del vettore
 *
 * @param vector - puntatore al vettore da settare ad 1
 * @param size - la dimensione del vettore
 */
__global__ void negateVector(TYPE *vector, int size)
{
    for (int idX = threadIdx.x + blockIdx.x * blockDim.x;
         idX < size;
         idX += gridDim.x * blockDim.x)
    {
        vector[idX] = -vector[idX];
    }
}

/*
 * Genera in parallelo il vettore della soluzione nella memoria device
 */
__global__ void getSolution(TYPE *source, int *base, int baseSize, TYPE *out, int lastVar)
{
    for (int idX = threadIdx.x + blockIdx.x * blockDim.x;
         idX < baseSize;
         idX += gridDim.x * blockDim.x)
    {
        int var = base[idX];
        if (var < lastVar)
        {
            out[var] = source[idX];
        }
    }
}

/*
 * Dato un valore minimo ed un valore massimo controlla se nel vettore in input ce ne è uno compreso
 * (min <= x < max)
 */
__global__ void checkVector(int *vector, int size, int min, int max, int *out)
{
    for (int idX = threadIdx.x + blockIdx.x * blockDim.x;
         idX < size;
         idX += gridDim.x * blockDim.x)
    {
        if (vector[idX] < max && vector[idX] >= min)
        {
            atomicAdd(out, 1);
        }
    }
}

void fillTableu(tabular_t *tabular, int *base)
{
    hipStream_t streams[6];
    for (size_t i = 0; i < 6; i++)
        HANDLE_ERROR(hipStreamCreate(streams + i));

    // Punto 1: primi n + m valori a 0 della funzione dei costi
    int sizeToSetZero = tabular->problem->vars + tabular->problem->constraints + 1;
    HANDLE_ERROR(hipMemsetAsync(tabular->costsVector, 0, BYTE_SIZE(sizeToSetZero), streams[0]));

    // Punto 2: ultimi m valori della prima riga a 1 (kernel)
    setVectorToOne<<<BL(tabular->problem->constraints), THREADS, 0, streams[1]>>>(tabular->costsVector + sizeToSetZero, tabular->problem->constraints);

    // Punto 3: copia della matrice dei vincoli originale dalla seconda riga di tabular->table sulle prime n colonne (hipMemcpy2DAsync)
    HANDLE_ERROR(hipMemcpy2DAsync(
        tabular->constraintsMatrix,               // destinazione
        tabular->pitch,                           // pitch della destinazione
        tabular->problem->constraintsMatrix,      // fonte
        BYTE_SIZE(tabular->problem->constraints), // pitch della fonte
        BYTE_SIZE(tabular->problem->constraints), // larghezza matrice
        tabular->problem->vars,                   // altezza matrice
        hipMemcpyDefault,                        // tipo
        streams[2]                                // stream
        ));

    // Punto 4 e 5: riempimento delle successive m colonne con identità (kernel)
    fillMatrix<<<BL(tabular->cols), THREADS, 0, streams[3]>>>(
        ROW(tabular->constraintsMatrix, tabular->problem->vars, tabular->pitch),
        tabular->cols, // colonne della matrice
        tabular->pitch // pitch
    );

    // Punto 6: copia del vettore dei termini noti nel vettore degli indicatori (hipMemcpyAsync)
    HANDLE_ERROR(hipMemcpyAsync(
        tabular->knownTermsVector,                // puntatore al vettore destinazione (vettore indicatori)
        tabular->problem->knownTermsVector,       // fonte
        BYTE_SIZE(tabular->problem->constraints), // dimensione in byte del vettore
        hipMemcpyDefault,                        // tipo
        streams[4]                                // stream
        ));

    // Punto 7: riempimento vettore della base con numeri progressivi da n+m a n+2m-1 (kernel)
    fillBaseVector<<<BL(tabular->cols), THREADS, 0, streams[5]>>>(
        base,
        tabular->cols,
        tabular->problem->vars + tabular->problem->constraints);

    HANDLE_KERNEL_ERROR();

    for (size_t i = 0; i < 6; i++)
        HANDLE_ERROR(hipStreamDestroy(streams[i]));
}

// TODO: esiste una versione migliore?
/**
 * Controlla se il problema è degenere
 * @return DEGENERATE se degenere, FEASIBLE altrimenti
 */
int checkIfDegenerate(tabular_t *tabular, int *base)
{
    int *checkDegenere_h, *checkDegenere_map;
    HANDLE_ERROR(hipHostAlloc(&checkDegenere_h, sizeof(int), hipHostMallocMapped));
    HANDLE_ERROR(hipHostGetDevicePointer(&checkDegenere_map, checkDegenere_h, 0));
    HANDLE_ERROR(hipMemset(checkDegenere_map, 0, sizeof(int)));

    checkVector<<<BL(tabular->cols), THREADS>>>(
        base,
        tabular->cols,
        tabular->problem->vars + tabular->cols,
        tabular->problem->vars + 2 * tabular->cols,
        checkDegenere_map);
    HANDLE_KERNEL_ERROR();

    int checkDegenere = *checkDegenere_h;
    HANDLE_ERROR(hipHostFree(checkDegenere_h));

    if (checkDegenere > 0)
        return DEGENERATE;
    else
        return FEASIBLE;
}

int phase1(tabular_t *tabular, int *base_h, int *base_dev)
{
    // Fase 1: riempimento del tableu
    fillTableu(tabular, base_dev);
#ifdef DEBUG
    fprintf(stdout, "\nTableu nella situazione iniziale\n");
    printTableauToStream(stdout, tabular, base_h);
#endif

    // Fase 2: eliminazione di gauss
    updateObjectiveFunction(tabular, base_dev);
#ifdef DEBUG
    fprintf(stdout, "\nTableu dopo l'eliminazione di gauss\n");
    printTableauToStream(stdout, tabular, base_h);
#endif

    // Fase 3: lancio del solver
    solve(tabular, base_h);
#ifdef DEBUG
    fprintf(stdout, "\nTableu dopo il lancio del primo solver\n");
    printTableauToStream(stdout, tabular, base_h);
#endif

    // Fase 4: controllo infattibilità
    TYPE firstKnownTermsValue;
    HANDLE_ERROR(hipMemcpy(&firstKnownTermsValue, tabular->costsVector, BYTE_SIZE(1), hipMemcpyDeviceToHost));
    if (firstKnownTermsValue < 0)
        return INFEASIBLE;

    // Fase 5: controllo degenere: se è presente in base un valore x tale che n+m <= x < n+2m, il problema è degenere
    return checkIfDegenerate(tabular, base_dev);
}

int phase2(tabular_t *tabular, int *base_h, int *base_dev)
{
    // Fase 1: riduzione del numero di colonne
    tabular->rows -= tabular->cols;

#ifdef DEBUG
    fprintf(stdout, "\nTableu dopo aggiornamento colonne in phase2\n");
    printTableauToStream(stdout, tabular, base_h);
#endif

    // Fase 2: riempimento vettore costi su due stream diversi
    hipStream_t streams[2];
    for (size_t i = 0; i < 2; i++)
        HANDLE_ERROR(hipStreamCreate(streams + i));

    // ultimi m elementi a 0
    HANDLE_ERROR(hipMemsetAsync(
        1 + tabular->costsVector + tabular->problem->vars,
        0,
        BYTE_SIZE(tabular->cols),
        streams[0]));

    HANDLE_ERROR(hipMemcpyAsync(
        tabular->costsVector + 1,
        tabular->problem->objectiveFunction,
        BYTE_SIZE(tabular->problem->vars),
        hipMemcpyDefault,
        streams[1]));
    negateVector<<<BL(tabular->problem->vars), THREADS, 0, streams[1]>>>
        (tabular->costsVector + 1, tabular->problem->vars);
    HANDLE_KERNEL_ERROR();

    for (size_t i = 0; i < 2; i++)
        HANDLE_ERROR(hipStreamDestroy(streams[i]));

#ifdef DEBUG
    fprintf(stdout, "\nTableu dopo riempimento funzione obiettivo in phase2\n");
    printTableauToStream(stdout, tabular, base_h);
#endif

    // Fase 3: Eliminazione di gauss per esprimere la funzione obiettivo in termini delle variabili non di base
    updateObjectiveFunction(tabular, base_dev);
#ifdef DEBUG
    fprintf(stdout, "\nTableu dopo eliminazione di gauss in phase2\n");
    printTableauToStream(stdout, tabular, base_h);
#endif

// Fase 4: Esecuzione dell'algoritmo di risoluzione
#ifdef DEBUG
    int esito = solve(tabular, base_h);
    fprintf(stdout, "\nTableu dopo seconda esecuzione del solver\n");
    printTableauToStream(stdout, tabular, base_h);
    return esito;
#else
    return solve(tabular, base_h);
#endif
}

__inline__ void unregisterMemory(int *base_h, problem_t *problem)
{
    HANDLE_ERROR(hipHostUnregister(problem->constraintsMatrix));
    HANDLE_ERROR(hipHostUnregister(problem->knownTermsVector));
    HANDLE_ERROR(hipHostUnregister(problem->objectiveFunction));
    HANDLE_ERROR(hipHostFree(base_h));
}

void getSolutionHost(tabular_t *tabular, int *base, TYPE *solution, TYPE *optimalValue)
{
    HANDLE_ERROR(hipMemcpy(optimalValue, tabular->costsVector, BYTE_SIZE(1), hipMemcpyDefault));

    TYPE *dev_solution;
    HANDLE_ERROR(hipHostRegister(solution, BYTE_SIZE(tabular->problem->vars), hipHostRegisterMapped));
    HANDLE_ERROR(hipHostGetDevicePointer(&dev_solution, solution, 0));
    HANDLE_ERROR(hipMemset(dev_solution, 0, BYTE_SIZE(tabular->problem->vars)));

    getSolution<<<BL(tabular->cols), THREADS>>>
        (tabular->knownTermsVector, base, tabular->cols, dev_solution, tabular->problem->vars);
    HANDLE_KERNEL_ERROR();

    HANDLE_ERROR(hipHostUnregister(solution));
}

int twoPhaseMethod(problem_t *problem, TYPE *solution, TYPE *optimalValue)
{
    tabular_t *tabular = newTabular(problem);
    int *base_h;
    int *base_map;

    // Uso memoria di tipo mapped per memorizzare il vettore di base
    HANDLE_ERROR(hipHostAlloc(&base_h, tabular->cols * sizeof(int), hipHostMallocMapped)); // il vettore della base ha dimensione vettore dei vincoli => tabular->cols
    HANDLE_ERROR(hipHostGetDevicePointer(&base_map, base_h, 0));

    // Registro i vettori del problema come memoria page-locked (per poter utilizzare i trasferimenti paralleli con gli stream)
    HANDLE_ERROR(hipHostRegister(problem->constraintsMatrix, BYTE_SIZE(problem->vars * problem->constraints), hipHostRegisterDefault));
    HANDLE_ERROR(hipHostRegister(problem->knownTermsVector, BYTE_SIZE(problem->constraints), hipHostRegisterDefault));
    HANDLE_ERROR(hipHostRegister(problem->objectiveFunction, BYTE_SIZE(problem->vars), hipHostRegisterDefault));

    int result = phase1(tabular, base_h, base_map);
    if (result != FEASIBLE)
    {
        unregisterMemory(base_h, problem);
        return result;
    }

    result = phase2(tabular, base_h, base_map);
    if (result != FEASIBLE)
    {
        unregisterMemory(base_h, problem);
        return result;
    }

    getSolutionHost(tabular, base_map, solution, optimalValue);

    unregisterMemory(base_h, problem);
    return result;
}