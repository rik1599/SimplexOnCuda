﻿#include "error.cuh"
#include <stdio.h>
#include <stdlib.h>

void HandleError(hipError_t err, const char * file, int line)
{
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

void checkKernelError(const char * file, int line)
{
	hipDeviceSynchronize();
	HandleError(hipGetLastError(), file, line);
}
