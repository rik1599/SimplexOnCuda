#include "hip/hip_runtime.h"
#include "solver.h"
#include "twoPhaseMethod.h"
#include "reduction.cuh"
#include "error.cuh"

struct matrixInfo
{
    TYPE *mat;
    size_t pitch;
    int rows;
    int cols;
};

#define TILE_DIM 32
// (N - TILE_DIM)/(8192 - TILE_DIM) = (BLOCK_DIM(N) - 1)/15
#define BLOCK_DIM(N) ceil((N + 512.0) / 544.0)

#define THREADS 512
#define BL(N) min((N + THREADS - 1) / THREADS, 1024)

/** Copio una colonna della matrice (con accesso strided) in un vettore in memoria globale
 */
__global__ void copyColumn(matrixInfo matInfo, int colToCpy, TYPE *dst)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;

    for (; i < matInfo.rows; i += step)
    {
        dst[i] = ROW(matInfo.mat, i, matInfo.pitch)[colToCpy];
    }
}

/** 9) Aggiornamento tableau per tile. Per ogni A[y][x]
 *      se y == colPivotIndex allora A[y][x] = A[y][x] * recPivot
 *      altrimenti  A[y][x] = - colPivot[y] * rowPivot[x] * recPivot + A[y][x]
 */
__global__ void updateVariables(matrixInfo matInfo, double *colPivot, double *rowPivot, int colPivotIndex, double pivot)
{
    // coordinate
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // dimensioni griglia
    int nx = blockDim.x * gridDim.x;
    int ny = blockDim.y * gridDim.y;

    double *pRow;
    char *pMat = (char *)matInfo.mat;
    for (int col = x; col < matInfo.cols; col += nx)
    {
        for (int row = y; row < matInfo.rows; row += ny)
        {
            pRow = (double *)(pMat + row * matInfo.pitch);

            pRow[col] = col == colPivotIndex ? __drcp_rd(pivot) * pRow[col] : pRow[col] - (rowPivot[col] * __drcp_rd(pivot) * colPivot[row]);
        }
    }
}

__global__ void updateCostsVector(TYPE* costVector, int size, double *colPivot, double costsPivot, double pivot)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;

    for (; i < size; i += step)
    {
        costVector[i] = costVector[i] - (costsPivot * __drcp_rd(pivot) * colPivot[i]);
    }
}

__inline__ void updateAll(tabular_t *tabular, TYPE *colPivot, int colPivotIndex, TYPE *rowPivot, TYPE minCosts)
{
    matrixInfo matInfo = {tabular->table, tabular->pitch, tabular->rows, tabular->cols};

    copyColumn<<<BL(tabular->rows), THREADS>>>(matInfo, colPivotIndex, colPivot);
    HANDLE_KERNEL_ERROR();

    TYPE pivot;
    HANDLE_ERROR(hipMemcpy(&pivot, rowPivot + colPivotIndex, BYTE_SIZE(1), hipMemcpyDefault));

    hipStream_t streams[2];
    dim3 block(TILE_DIM, TILE_DIM);
    dim3 grid(BLOCK_DIM(tabular->cols), BLOCK_DIM(tabular->rows));
    hipStreamCreate(&streams[0]);
    updateVariables<<<grid, block, 0, streams[0]>>>(matInfo, colPivot, rowPivot, colPivotIndex, pivot);

    hipStreamCreate(&streams[1]);
    updateCostsVector<<<BL(tabular->rows), THREADS, 0, streams[1]>>>(tabular->costsVector, tabular->rows, colPivot, minCosts, pivot);

    HANDLE_KERNEL_ERROR();

    hipStreamDestroy(streams[0]);
    hipStreamDestroy(streams[1]);
}

int solve(tabular_t *tabular, int *base)
{
    TYPE *rowPivot, *colPivot;
    HANDLE_ERROR(hipMalloc((void **)&rowPivot, BYTE_SIZE(tabular->cols)));
    HANDLE_ERROR(hipMalloc((void **)&colPivot, BYTE_SIZE(tabular->rows)));

    unsigned int colPivotIndex;
    unsigned int rowPivotIndex;
    TYPE minCosts;

    while ((minCosts = minElement(tabular->costsVector + 1, tabular->rows - 1, &rowPivotIndex)) < 0)
    {
        HANDLE_ERROR(hipMemcpy(rowPivot, ROW(tabular->constraintsMatrix, rowPivotIndex, tabular->pitch), BYTE_SIZE(tabular->cols), hipMemcpyDefault));

        if (isLessThanZero(rowPivot, tabular->cols))
        {
            return UNBOUNDED;
        }

        minElement(tabular->indicatorsVector, rowPivot, tabular->cols, &colPivotIndex);
        
        base[colPivotIndex] = rowPivotIndex; 

        updateAll(tabular, colPivot, colPivotIndex, rowPivot, minCosts);
    }

    HANDLE_ERROR(hipFree(rowPivot));
    HANDLE_ERROR(hipFree(colPivot));
    return FEASIBLE;
}