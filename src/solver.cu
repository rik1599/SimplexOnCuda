#include "hip/hip_runtime.h"
#include "solver.h"
#include "twoPhaseMethod.h"
#include "reduction.cuh"
#include "error.cuh"

struct matrixInfo
{
    TYPE *mat;
    size_t pitch;
    int rows;
    int cols;
};

#define TILE_DIM 32
// (N - TILE_DIM)/(8192 - TILE_DIM) = (BLOCK_DIM(N) - 1)/15
#define BLOCK_DIM(N) ceil((N + 512.0) / 544.0)

#define THREADS 512
#define BL(N) min((N + THREADS - 1) / THREADS, 1024)

/** Copio una colonna della matrice (con accesso strided) in un vettore in memoria globale
 */
__global__ void copyColumn(matrixInfo matInfo, int colToCpy, TYPE *dst)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    char *pMat = (char *)matInfo.mat;

    for (; i < matInfo.rows; i += step)
    {
        dst[i] = ((TYPE *)(pMat + i * matInfo.pitch))[colToCpy];
    }
}

/** Costruisce il vettore degli indicatori lastCol/colPivot */
__global__ void createIndicatorVector(matrixInfo matInfo, int colPivotIndex)
{
}

/** 9) Aggiornamento tableau per tile. Per ogni A[y][x]
 *      se y == rowPivotIndex allora A[y][x] = A[y][x] * recPivot
 *      altrimenti  A[y][x] = - colPivot[y] * rowPivot[x] * recPivot + A[y][x]
 */
__global__ void updateMatrix(matrixInfo matInfo, double *colPivot, double *rowPivot, int colPivotIndex, double pivot)
{
    // coordinate
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // dimensioni griglia
    int nx = blockDim.x * gridDim.x;
    int ny = blockDim.y * gridDim.y;

    double *pRow;
    char *pMat = (char *)matInfo.mat;
    for (int col = x; col < matInfo.cols; col += nx)
    {
        for (int row = y; row < matInfo.rows; row += ny)
        {
            pRow = (double *)(pMat + row * matInfo.pitch);

            pRow[col] = col == colPivotIndex ? __drcp_rd(pivot) * pRow[col] : pRow[col] - (rowPivot[col] * __drcp_rd(pivot) * colPivot[row]);
        }
    }
}

int solve(tabular_t *tabular, int *base)
{
    TYPE *rowPivot, *colPivot;
    HANDLE_ERROR(hipMalloc((void **)&rowPivot, BYTE_SIZE(tabular->cols)));
    HANDLE_ERROR(hipMalloc((void **)&colPivot, BYTE_SIZE(tabular->rows)));

    unsigned int colPivotIndex;
    unsigned int rowPivotIndex;

    while (minElement(tabular->costsVector + 1, tabular->cols - 1, &rowPivotIndex) < 0)
    {
        rowPivotIndex++;

        double *pRowPivot = (double *)((char *)tabular->constraintsMatrix + rowPivotIndex * tabular->pitch);
        double pivot;

        HANDLE_ERROR(hipMemcpy(rowPivot, pRowPivot, BYTE_SIZE(tabular->cols), hipMemcpyDefault));

        if (isLessThanZero(rowPivot, tabular->cols))
        {
            return UNBOUNDED;
        }

        HANDLE_ERROR(hipMemcpy(&pivot, pRowPivot + colPivotIndex, BYTE_SIZE(1), hipMemcpyDefault));

        matrixInfo matInfo = {tabular->table, tabular->pitch, tabular->rows, tabular->cols};

        copyColumn<<<BL(tabular->rows), THREADS>>>(matInfo, colPivotIndex, colPivot);
        HANDLE_KERNEL_ERROR();

        dim3 block(TILE_DIM, TILE_DIM);
        dim3 grid(BLOCK_DIM(tabular->cols), BLOCK_DIM(tabular->rows));
        updateMatrix<<<grid, block>>>(matInfo, colPivot, rowPivot, colPivotIndex, pivot);
        HANDLE_KERNEL_ERROR();
    }

    HANDLE_ERROR(hipFree(rowPivot));
    HANDLE_ERROR(hipFree(colPivot));
    return FEASIBLE;
}